#include "hip/hip_runtime.h"
//
//  main.cpp
//  High End simulation in Practice
//
//  Created by Sagar Dolas on 19/04/16.
//  Copyright © 2016 Sagar Dolas. All rights reserved.
//

#include <hip/hip_runtime.h>
#include <cstddef>
#include <sys/time.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <string>
#include "lodepng.h"

//#define GRID_SIZE = 1<<11 ; 

void checkError (hipError_t err)
{
    if(err != hipSuccess )
    {
        std::cout<< hipGetErrorString(err) <<std::endl ;
        exit(-1);
    }
}
double getSeconds()
{
    struct timeval tp ;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6) ;
}

// Device Function //

__global__ void juliaImage(unsigned char * color_bit_device, long long N, const double mesh, const double threshold) {
    
    long long idx = blockIdx.x * blockDim.x + threadIdx.x ;
    double real = (mesh * (idx % N)) - 2.0;
    double imag = (mesh * (idx / N)) - 2.0 ;
    double temp_real = 0.0 ; 
    double temp_imag = 0.0 ; 
    const double c_real = -0.4 ; 
    const double c_imag = 0.6 ;
    double modulus = sqrt( (real * real) + (imag * imag) ) ;     
    unsigned int itr = 0 ; 
    
    while (modulus <= threshold) { 
	
      temp_real = (real * real) - (imag * imag) ; 
      temp_imag = (2 * real * imag) ; 
      
      real = temp_real ; 
      imag = temp_imag ; 
      
      real += c_real ;
      imag += c_imag ; 
      
      temp_real = 0.0 ; 
      temp_imag = 0.0 ; 
      
      modulus = sqrt( (real * real) + (imag * imag) ) ; 
      
      ++itr ;  
    }
    unsigned int numIter_ = itr*10 +10000;
    unsigned i = idx % N ;
    unsigned j = idx / N ;

    color_bit_device[4*N*j+4*i+3] = (numIter_  & 255);
    color_bit_device[4*N*j+4*i+2] = (numIter_ >> 8) & 255;
    color_bit_device[4*N*j+4*i+1] = (numIter_ >> 16) & 255;
    color_bit_device[4*N*j+4*i+0] = (numIter_ >> 24) & 255;

}

//Encode from raw pixels to disk with a single function call
//The image argument has width * height RGBA pixels or width * height * 4 bytes
void encodeImage(const char* filename, std::vector<unsigned char>& image, unsigned width, unsigned height)
{
  //Encode the image
  unsigned error = lodepng::encode(filename, image, width, height);
  //if there's an error, display it
  if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;
}

int main(int argc , char *argv[]) {


    unsigned int numTHreadsPerBlocks_ = 1024;         //std::stol(argv[1]);
    unsigned numblocks_ =  2048 *2048 / numTHreadsPerBlocks_ ;

    const long long numberOfGridPoints_ = (1<<11)* (1<<11 ) ;
    const long long bytes_ = numberOfGridPoints_ * sizeof(unsigned char ) * 4 ;
    const long num = 2048 ;  
    const double mesh = 4.0 / ((1<<11 )) ;
    const double threshold = 500.0 ; 
    
    std::cout<<"the mesh is "<<mesh<<std::endl ; 
    std::cout<<"The Total Memory in MB allocated for the program is :="<<bytes_ * 1e-6<<std::endl ;
    
    // Allocating Vector on Host
    std::vector <unsigned char> colourBit(numberOfGridPoints_*4);

    // Pointer on device
    unsigned char * color_bit_device  ;
    
    // Allocating memory on device
    checkError(hipMalloc(&color_bit_device,bytes_)) ;
     
    double start = getSeconds() ;
    juliaImage<<< numblocks_ ,numTHreadsPerBlocks_>>>(color_bit_device,num,mesh,threshold) ;
    checkError(hipDeviceSynchronize());
    double end = getSeconds() ;
    std::cout<< (end - start) *1e3 <<std::endl ;
    
    // Copying data back to Host
    checkError(hipMemcpy(&colourBit[0],color_bit_device,bytes_,hipMemcpyDeviceToHost));

    // Freeing the memory on the device
    checkError(hipFree(color_bit_device)) ;
      
    encodeImage("Julia.png", colourBit, num, num);    
    return 0 ;
}
