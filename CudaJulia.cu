#include "hip/hip_runtime.h"
//
//  main.cpp
//  High End simulation in Practice
//
//  Created by Sagar Dolas on 19/04/16.
//  Copyright © 2016 Sagar Dolas. All rights reserved.
//

#include <hip/hip_runtime.h>
#include <cstddef>
#include <sys/time.h>
#include <iostream>
#include <vector>
#include <cmath>
#include "lodepng.h"

//#define GRID_SIZE = 1<<11 ; 

void checkError (hipError_t err)
{
    if(err != hipSuccess )
    {
        std::cout<< hipGetErrorString(err) <<std::endl ;
        exit(-1);
    }
}
double getSeconds()
{
    struct timeval tp ;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6) ;
}

class Complex {
    
    private :
    
    double real ;
    double imag ;
    
public:
    
    __host__ __device__ Complex(const double _real, const double _imag): real(_real), imag(_imag){}
    __host__ __device__ Complex(){
        this->real = 0.0 ;
        this->imag = 0.0 ;
    }
    __host__ __device__ ~Complex(){} ;
    
    // Access for the real part
    
    __host__ __device__ const double& realpart() const {
        return this->real ;
    }
    
    __host__ __device__ double & realpart() {
        return this->real ;
    }
    
    // Access for the imag part
    
    __host__ __device__ const double & imagpart() const {
        return this->imag ;
    }
    
    __host__ __device__ double & imagpart() {
        return this->imag ;
    }
    
    __host__ __device__ const Complex square() {
        Complex temp ;
        
        temp.realpart() = (this->realpart() * this->realpart()) - (this->imagpart() * this->imagpart()) ;
        temp.imagpart() = (2 * this->realpart() * this->imagpart() ) ;
        
        this->realpart() = temp.realpart() ;
        this->imagpart() = temp.imagpart() ;
        
        return *(this) ;
        
    }
    
    __host__ __device__ const double  modulus() const{
        return std::sqrt((this->real * this->real) + (this->imag * this->imag)) ;
    }
    
    __host__ __device__ Complex operator+ (const Complex & obj){
        Complex temp ;
        
        double real = this->real + obj.realpart() ;
        double imag = this->imag + obj.imagpart() ;
        
        return Complex(real,imag) ;
    }
    
    __host__ __device__ Complex& operator= (const Complex & obj){
        this->real = obj.realpart() ;
        this->imag = obj.imagpart() ;
        
        return *(this) ;
    }
};

// Device Function //

__global__ void juliaImage(double * color_bit_device, long long N, const double mesh) {
    
    long long idx = blockIdx.x * blockDim.x + threadIdx.x ;
    double real = (mesh * (idx % N)) - 2.0;
    double imag = (mesh * (idx / N)) - 2.0 ;
    const int numIter = 25 ;
    double temp_real = 0.0 ; 
    double temp_imag = 0.0 ; 
    const double c_real = -0.4 ; 
    const double c_imag = 0.6 ;
    
    
    for(size_t i =0 ; i < numIter; ++i){
	
      temp_real = (real * real) - (imag * imag) ; 
      temp_imag = (2 * real * imag) ; 
      
      real = temp_real ; 
      imag = temp_imag ; 
      
      real += c_real ; 
      imag += c_imag ; 
      
      temp_real = 0.0 ; 
      temp_imag = 0.0 ; 
    }
    
    double modulus = sqrt( (real * real) + (imag * imag) ) ; 
    color_bit_device[idx] = modulus ; 
}

//Encode from raw pixels to disk with a single function call
//The image argument has width * height RGBA pixels or width * height * 4 bytes
void encodeImage(const char* filename, std::vector<unsigned char>& image, unsigned width, unsigned height)
{
  //Encode the image
  unsigned error = lodepng::encode(filename, image, width, height);

  //if there's an error, display it
  if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;
}


int main() {
    
    
    const long long numberOfGridPoints_ = (1<<11)* (1<<11 ) ;
    const long long bytes_ = numberOfGridPoints_ * sizeof(double) ;
    const long num = 2048 ;  
    const double mesh = 4.0 / ((1<<11 )) ;
    
    std::cout<<"the mesh is "<<mesh<<std::endl ; 
    std::cout<<"The Total Memory in MB allocated for the program is :="<<bytes_ * 1e-6<<std::endl ;
    
    // Allocating Vector on Host
    std::vector<double> color_bit(numberOfGridPoints_,0) ;
    std::vector <unsigned char> colourBit(numberOfGridPoints_*4);

    
    // Pointer on device
    double * color_bit_device  ;
    
    // Allocating memory on device
    checkError(hipMalloc(&color_bit_device,bytes_)) ;
    
    // Copying Data from host to device
    checkError(hipMemcpy(color_bit_device,&color_bit[0],bytes_,hipMemcpyHostToDevice)) ;
    
    double start = getSeconds() ;
    juliaImage<<<(1<<12),(1<<10)>>>(color_bit_device,num,mesh) ;
    double end = getSeconds() ;
    
    hipDeviceSynchronize() ; 
    // Copying data back to Host
    checkError(hipMemcpy(&color_bit[0],color_bit_device,bytes_,hipMemcpyDeviceToHost));
    
    /*
    for(unsigned int i =0 ; i < numberOfGridPoints_ ; ++i ){
	
      // checking if the value is edited or not 
      std::cout<<color_bit[i]<<std::endl ;
      
    }
    */
    // Freeing the memory on the device
    checkError(hipFree(color_bit_device)) ;
   
    //Generation of Image
      for(int j=0;j<num;j++){
        for(int i=0;i<num;i++){
            //std::cout<<moduli[j*num+i]<<std::endl;
            if(color_bit[j*num+i] > 10.0){
       
                colourBit[4*num*j+4*i+0] = 255;
                colourBit[4*num*j+4*i+1] = 255;
                colourBit[4*num*j+4*i+2] = 0;
                colourBit[4*num*j+4*i+3] = 0;
            }
            else{
      
                colourBit[4*num*j+4*i+0] = 0;
                colourBit[4*num*j+4*i+1] = 0;
                colourBit[4*num*j+4*i+2] = 255;
                colourBit[4*num*j+4*i+3] = 255;
            }
        }
    }
      
    encodeImage("Julia.png", colourBit, num, num);    
    return 0 ;
}

